#include "hip/hip_runtime.h"
//
// Created by The MAC PRO on 2020/7/9.
//

#include "../include/testAligner.cuh"

int main(int argc, char **argv) {
    srand(time(NULL));
    ProfilerStart("testAligner.prof");
    if (argc < 2) {
        std::cout << "Usage ./testAligner filename" << std::endl;
        return 1;
    }
    TestAligner ta(argv[1]);
    size_t k, kMerNumTh, baseOverlapTh, n;
    while (true) {
        std::cout << "k kMerNumth baseOverlapTh n" << std::endl;
        std::cin >> k >> kMerNumTh >> baseOverlapTh >> n;
        if (k == 0)
            break;
        for (size_t i = 0; i < n; ++i)
            ta.test(k, kMerNumTh, baseOverlapTh);
    }
    ProfilerStop();
}

TestAligner::TestAligner(const char *fileName) : nR(fileName, 1, 1) {
    std::cout << "Finished Initializing TestAligner from " << fileName << std::endl;
}

void TestAligner::test(const size_t k, const size_t kMerNumTh, const size_t baseOverlapTh) {
    ReadAligner *rA = new MergeSortReadAligner(k, kMerNumTh);
    size_t randomReadIndex = rand() % nR.numReads;
    const std::string &randomRead = *nR.readData[randomReadIndex];
    const long randomPos = nR.readPos[randomReadIndex];
    const long th = nR.readLen - baseOverlapTh;
    size_t numPositives = 0;
    size_t truePositives = 0;
    size_t falsePositives = 0;
    size_t numNegatives = 0;
    size_t falseNegatives = 0;
    size_t trueNegatives = 0;
    double posError = 0;
    for (size_t i = 0; i < nR.numReads; ++i) {
        if (i == randomReadIndex)
            continue;
//        if (i % 1000 == 0)
//            std::cout << i << std::endl;
        ssize_t relPos;
        if (rA->align(randomRead, *nR.readData[i], relPos)) {
            //std::cout << "Real " << (long) nR.readPos[i] - (long) randomPos
            //<< " Predicted " << relPos << std::endl;
            posError += abs((long) nR.readPos[i] - (long) randomPos
                            - relPos);
            numPositives++;
            if (abs(randomPos - (long) nR.readPos[i]) > th) {
                falsePositives++;
                //std::cout << (long) nR.readLen - abs((long) randomPos - (long) nR.readPos[i]) << std::endl;
            } else
                truePositives++;
        } else {
            numNegatives++;
            if (abs(randomPos - (long) nR.readPos[i]) > th)
                trueNegatives++;
            else
                falseNegatives++;
        }

    }
    posError /= numPositives;
    std::cout << "Average position error is " << posError << std::endl;
    const int w = 13;
    std::cout << std::setw(w) << "k" << ","
              << std::setw(w) << "kMerNumTh" << ","
              << std::setw(w) << "baseOverlapTh" << ","
              << std::setw(w) << "totalPos" << ","
              << std::setw(w) << "totalNeg" << ","
              << std::setw(w) << "falsePos" << ","
              << std::setw(w) << "falseNeg" << ","
              << std::setw(w) << "numOverlaps" << std::endl;
    std::cout << std::setw(w) << k << ","
              << std::setw(w) << kMerNumTh << ","
              << std::setw(w) << baseOverlapTh << ","
              << std::setw(w) << numPositives << ","
              << std::setw(w) << numNegatives << ","
              << std::setw(w) << falsePositives << ","
              << std::setw(w) << falseNegatives << ","
              << std::setw(w) << truePositives + falseNegatives << std::endl;
    delete rA;
}
