#include "hip/hip_runtime.h"
#include "../include/NanoporeReads.cuh"

NanoporeReads::NanoporeReads(char *fileName, int k, int n) : k(k), n(n), sketches(NULL) {
    std::ifstream infile(fileName);
    std::string line;
    numReads = 0;
    while (std::getline(infile, line)) {
//        std::cout << line << std::endl;
        size_t index = line.find(':');
        readPos.push_back(std::stol(line.substr(0, index)));
        {
            std::unique_ptr<std::string> ptr(new std::string(line.substr(index + 1)));
            editStrings.push_back(std::move(ptr));
        }
//        std::cout << readPos.back() << std::endl;
//        std::cout << editStrings.back() << std::endl;
        std::getline(infile, line);
        {
            std::unique_ptr<std::string> ptr(new std::string(line));
            readData.push_back(std::move(ptr));
        }
        numReads++;
    }
    readLen = readData[0]->length();
    readPosSorted = readPos;
    std::sort(readPosSorted.begin(), readPosSorted.end());
    std::cout << "numReads " << numReads << std::endl;
    std::cout << "readLen " << readLen << std::endl;
}

void NanoporeReads::calculateMinHashSketches() {
    // We store all the k-mers as uint64s. This would work for all k<=32,
    // which is definitely sufficient
    const size_t numKMers = readLen - k + 1;
    kMer_t *kMers;
    // Because of memory constraints on the GPUs we cannot deal with all the reads at once.
    // So we arrange the reads into blocks of blockSize reads and only work on a single block
    // at the same time.
    const size_t blockSize = 2048;
    std::cout << "numKMers " << numKMers << std::endl;

    hipMallocManaged(&(sketches), n * numReads * sizeof(kMer_t));

    std::random_device rd;
    std::mt19937_64 gen(rd());

    /* This is where you define the number generator for unsigned long long: */
    std::uniform_int_distribution<unsigned long long> dis;

    kMer_t *randNumbers;
    hipMallocManaged(&randNumbers, n * sizeof(kMer_t));
    for (size_t i = 0; i < n; ++i) {
        randNumbers[i] = dis(gen);
    }

    for (size_t currentRead = 0; currentRead < numReads; currentRead += blockSize) {
        std::cout << "CurrentRead " << currentRead << std::endl;
        const long readsLeft = numReads - (long) currentRead;
        const size_t currentBlockSize = readsLeft > blockSize ? blockSize : readsLeft;

        auto generateKMers = [&]() {
            hipMallocManaged(&kMers, currentBlockSize * numKMers * sizeof(kMer_t));

            for (size_t i = 0; i < currentBlockSize; i++) {
                size_t baseIndex = i * numKMers;
#pragma omp parallel for
                for (size_t index = baseIndex; index < numKMers + baseIndex; index++) {
                    kMers[index] =
                            kMerToInt(readData[i + currentRead]->substr(
                                    index - baseIndex, k));
                }
            }
        };

        generateKMers();

//        for (size_t i = 0; i < currentBlockSize * numKMers; ++i) {
//            std::cout << i << " " << kMers[i] << std::endl;
//        }

        // Now we generate all hashes
        // hashes is indexed by (read number, k-mer number, hash number)
        kMer_t *hashes;
        hipMallocManaged(&hashes, n * currentBlockSize * numKMers * sizeof(kMer_t));

        const size_t blockSize = 512;
        const size_t numBlocks = 512;
        hashKMer <<< numBlocks, blockSize >>>(currentBlockSize * numKMers,
                                              n, kMers, hashes, randNumbers);
        // Finish calculating the hashes and frees unneeded memory
        hipDeviceSynchronize();

//        for (size_t i = 0; i < currentBlockSize * numKMers * n; ++i) {
//            std::cout << i << " " << hashes[i] << std::endl;
//        }

        // Now we are going to compute the sketches which are the minimums of the hashes
        calcSketch
        <<< (currentBlockSize + blockSize - 1)
        / blockSize, blockSize >>>(currentBlockSize,
                                   currentRead, numKMers,
                                   n, hashes,
                                   sketches, kMers);
        hipDeviceSynchronize();
        hipFree(kMers);
        hipFree(hashes);
    }
    hipFree(randNumbers);
    populateHashTables();
}

kMer_t NanoporeReads::kMerToInt(const std::string &s) {
    size_t l = s.length();
    kMer_t result = 0;
    for (size_t i = 0; i < l; ++i) {
        result <<= 2;
        result |= baseToInt(s[i]);
    }
    return result;
}

char NanoporeReads::baseToInt(const char base) {
    switch (base) {
        case 'A':
            return 0;
        case 'T':
            return 1;
        case 'C':
            return 2;
        case 'G':
            return 3;
        default:
            std::cout << "Oh No!" << std::endl;
            return 0;
    }
}

__global__ void hashKMer(const size_t totalKMers, const size_t n,
                         kMer_t *kMers, kMer_t *hashes, kMer_t *randNumbers) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    for (size_t i = index; i < totalKMers; i += stride) {
        size_t hashIndex = i * n;
        kMer_t currentHash = kMers[i];
        currentHash = (currentHash * (uint64_t) HASH_C64);
        currentHash ^= randNumbers[0];
        hashes[hashIndex++] = currentHash;
        for (size_t j = 1; j < n; j++) {
            kMer_t newHash = ((currentHash >> ROTATE_BITS)
                              | (currentHash << (KMER_BITS - ROTATE_BITS)))
                             ^0xABCD32108475AC38;
            newHash = (newHash * (uint64_t) HASH_C64);
            newHash ^= randNumbers[j];
            newHash += currentHash;
            currentHash = newHash;
            hashes[hashIndex++] = currentHash;
        }
    }
}

__global__ void calcSketch(const size_t numReads, const size_t currentRead,
                           const size_t numKMers, const size_t n,
                           kMer_t *hashes, kMer_t *sketches, kMer_t *kMers) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    for (size_t i = index; i < numReads; i += stride) {
        size_t sketchIndex = (i + currentRead) * n;
        for (size_t j = 0; j < n; ++j) {
            size_t hashIndex = i * n * numKMers + j;
            kMer_t currentMin = ~(kMer_t) 0;
            size_t minIndex = 0;
            for (size_t l = 0; l < numKMers; ++l) {
                kMer_t temp = hashes[hashIndex];
                hashIndex += n;
                minIndex = currentMin < temp ? minIndex : l;
                currentMin = currentMin < temp ? currentMin : temp;
            }
            //std::cout << "thread: " << i << " hash id: " << j << std::endl;
            //std::cout << "minIndex " << minIndex << ":" << kMers[i * numKMers + minIndex];

            if (kMers)
                sketches[sketchIndex++] = kMers[i * numKMers + minIndex];
            else
                sketches[sketchIndex++] = currentMin;
        }
    }
}

NanoporeReads::~NanoporeReads() {
    hipFree(sketches);
}

void NanoporeReads::printHashes() {
    for (size_t i = 0; i < numReads; ++i) {
        std::cout << readPos[i];
        for (size_t j = 0; j < n; ++j) {
            std::cout << ", \"" << sketches[i * n + j] << "\"";
        }
        std::cout << std::endl;
    }
}

void NanoporeReads::populateHashTables() {
    std::cout << "Starting to populate hash tables" << std::endl;
    auto start = std::chrono::high_resolution_clock::now();
    for (size_t i = 0; i < n; ++i) {
        hashTables.push_back(std::map<kMer_t, std::vector<size_t >>());
    }
//#pragma omp parallel for
    for (size_t i = 0; i < n; ++i) {
        std::map<kMer_t, std::vector<size_t>> &hT = hashTables[i];
        size_t currentIndex = i;
        for (size_t j = 0; j < numReads; ++j) {
            hT[sketches[currentIndex]].push_back(j);
//            std::cout << "Inserting " << j << " to " << sketches[currentIndex] << std::endl;
            currentIndex += n;
        }
    }

    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
    std::cout << "finished populating hash tables" << std::endl;
    std::cout << duration.count() << " milliseconds passed" << std::endl;
}

filterStats NanoporeReads::getFilterStats(unsigned int overlapBaseThreshold, unsigned int overlapSketchThreshold) {
    filterStats result(overlapBaseThreshold, overlapSketchThreshold);
    // First we calculate the number of overlaps and disjoints
    for (size_t i = 0; i < numReads; ++i) {
        long curTh = readPosSorted[i] + readLen - overlapBaseThreshold;
        for (size_t j = i + 1; j < numReads; ++j) {
            if (((long) readPosSorted[j]) <= curTh)
                result.numOverlaps++;
            else
                break;
        }
    }
    result.numDisjoint = (numReads * (unsigned long long) (numReads - 1)) / 2 - result.numOverlaps;

    // Now we calculate falsePositives, falseNegatives, etc
    for (size_t i = 0; i < numReads; ++i) {
        std::multiset<size_t> matches;
        unsigned long curPos = readPos[i];
        long th = readLen - overlapBaseThreshold;
        for (size_t sketchIndex = 0; sketchIndex < n; ++sketchIndex) {
            kMer_t curHash = sketches[i * n + sketchIndex];
//            std::cout << i << " " << sketchIndex << " " << curHash << std::endl;
//            auto currentMap = hashTables[sketchIndex];
//            for (auto p : currentMap) {
//                std::cout << p.first << " ";
//            }
//            std::cout << std::endl;
            std::vector<size_t> &m = hashTables[sketchIndex].at(curHash);
            matches.insert(m.begin(), m.end());
        }
        auto end = matches.end();
        for (auto it = matches.begin(); it != end; it = matches.upper_bound(*it)) {
            if (*it <= i)
                continue;
            unsigned long pos = readPos[*it];
            if (matches.count(*it) >= overlapSketchThreshold) {
                if (abs((long) pos - (long) curPos) > th)
                    result.falsePositives++;
                result.totalPositive++;
            }
        }
//        std::cout << std::endl;
    }

    result.totalNegative = result.numOverlaps + result.numDisjoint - result.totalPositive;
    result.falseNegatives = result.numOverlaps - result.totalPositive + result.falsePositives;
    return result;
}

filterStats::filterStats(unsigned int overlapBaseThreshold, unsigned int overlapSketchThreshold) :
        overlapBaseThreshold(overlapBaseThreshold), overlapSketchThreshold(overlapSketchThreshold),
        totalPositive(0), totalNegative(0), numOverlaps(0), numDisjoint(0),
        falsePositives(0), falseNegatives(0) {
}

std::ostream &operator<<(std::ostream &out, const filterStats &o) {
    const int w = 13;
    out << std::setw(w) << "overlapBaseTh" << ","
        << std::setw(w) << "numKMerTh" << ","
        << std::setw(w) << "totalPos" << ","
        << std::setw(w) << "totalNeg" << ","
        << std::setw(w) << "numOverlaps" << ","
        << std::setw(w) << "numDisjoint" << ","
        << std::setw(w) << "falsePos" << ","
        << std::setw(w) << "falseNeg" << std::endl;
    out << std::setw(w) << o.overlapBaseThreshold << ","
        << std::setw(w) << o.overlapSketchThreshold << ","
        << std::setw(w) << o.totalPositive << ","
        << std::setw(w) << o.totalNegative << ","
        << std::setw(w) << o.numOverlaps << ","
        << std::setw(w) << o.numDisjoint << ","
        << std::setw(w) << o.falsePositives << ","
        << std::setw(w) << o.falseNegatives << std::endl;
    return out;
}