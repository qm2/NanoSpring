#include "hip/hip_runtime.h"
#include "../include/NanoporeReads.cuh"

NanoporeReads::NanoporeReads(char *fileName, int k, int n) : k(k), n(n), sketches(NULL) {
    std::ifstream infile(fileName);
    std::string line;
    numReads = 0;
    while (std::getline(infile, line)) {
//        std::cout << line << std::endl;
        size_t index = line.find(':');
        this->readPos.push_back(std::stol(line.substr(0, index)));
        {
            std::unique_ptr <std::string> ptr(new std::string(line.substr(index + 1)));
            this->editStrings.push_back(std::move(ptr));
        }
//        std::cout << this->readPos.back() << std::endl;
//        std::cout << this->editStrings.back() << std::endl;
        std::getline(infile, line);
        {
            std::unique_ptr <std::string> ptr(new std::string(line));
            this->readData.push_back(std::move(ptr));
        }
        numReads++;
    }
    this->readLen = this->readData[0]->length();
    std::cout << "numReads " << numReads << std::endl;
    std::cout << "readLen " << readLen << std::endl;
}

void NanoporeReads::calculateMinHashSketches() {
    // We store all the k-mers as uint64s. This would work for all k<=32,
    // which is definitely sufficient
    const size_t numKMers = this->readLen - this->k + 1;
    kMer_t *kMers;
//    const size_t totalKMers = this->numReads * numKMers;
    const size_t blockSize = 2048;
    std::cout << "numKMers " << numKMers << std::endl;

    hipMallocManaged(&(this->sketches), this->n * this->numReads * sizeof(kMer_t));

    for (size_t currentRead = 0; currentRead < this->numReads; currentRead += blockSize) {
        std::cout << "CurrentRead " << currentRead << std::endl;
        const long readsLeft = numReads - (long) currentRead;
        const size_t currentBlockSize = readsLeft > blockSize ? blockSize : readsLeft;

        auto generateKMers = [&]() {
            hipMallocManaged(&kMers, currentBlockSize * numKMers * sizeof(kMer_t));

//#pragma omp parallel for collapse(2)
            for (size_t i = 0; i < currentBlockSize; i++) {
                size_t baseIndex = i * numKMers;
#pragma omp parallel for
                for (size_t index = baseIndex; index < numKMers + baseIndex; index++) {
                    kMers[index] =
                            kMerToInt(readData[i + currentRead]->substr(
                                    index - baseIndex, this->k));
                }
            }
        };

        generateKMers();

//        for (size_t i = 0; i < currentBlockSize * numKMers; ++i) {
//            std::cout << i << " " << kMers[i] << std::endl;
//        }

        // Now we generate all hashes
        // hashes is indexed by (read number, k-mer number, hash number)
        kMer_t *hashes;
        hipMallocManaged(&hashes, this->n * currentBlockSize * numKMers * sizeof(kMer_t));

        const size_t blockSize = 512;
        const size_t numBlocks = 512;
        hashKMer <<< numBlocks, blockSize >>>(currentBlockSize * numKMers,
                                              this->n, kMers, hashes);
        // Finish calculating the hashes and frees unneeded memory
        hipDeviceSynchronize();

        // Now we are going to compute the sketches which are the minimums of the hashes
        calcSketch
        <<< (currentBlockSize + blockSize - 1) / blockSize, blockSize >>>(currentBlockSize,
                                                                          currentRead, numKMers,
                                                                          this->n, hashes,
                                                                          this->sketches);
        hipDeviceSynchronize();
        hipFree(hashes);
    }
}

kMer_t NanoporeReads::kMerToInt(const std::string &s) {
    size_t l = s.length();
    kMer_t result = 0;
    for (size_t i = 0; i < l; ++i) {
        result <<= 2;
        result |= baseToInt(s[i]);
    }
    return result;
}

char NanoporeReads::baseToInt(const char base) {
    switch (base) {
        case 'A':
            return 0;
        case 'T':
            return 1;
        case 'C':
            return 2;
        case 'G':
            return 3;
        default:
            std::cout << "Oh No!" << std::endl;
            return 0;
    }
}

__global__ void hashKMer(const size_t totalKMers, const size_t n,
                         kMer_t *kMers, kMer_t *hashes) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    for (size_t i = index; i < totalKMers; i += stride) {
        size_t hashIndex = i * n;
        kMer_t currentHash = kMers[index];
        currentHash = (currentHash ^ (currentHash >> 30)) * UINT64_C(0xbf58476d1ce4e5b9);
        currentHash = (currentHash ^ (currentHash >> 27)) * UINT64_C(0x94d049bb133111eb);
        currentHash = currentHash ^ (currentHash >> 31);
        hashes[hashIndex++] = currentHash;
        for (size_t j = 1; j < n; j++) {
            currentHash = ((currentHash >> ROTATE_BITS)
                          | (currentHash << (KMER_BITS - ROTATE_BITS)))
                            ^ 0xABCD32108475AC38;
            hashes[hashIndex++] = currentHash;
        }
    }
}

__global__ void calcSketch(const size_t numReads, const size_t currentRead,
                           const size_t numKMers, const size_t n,
                           kMer_t *hashes, kMer_t *sketches) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    for (size_t i = index; i < numReads; i += stride) {
        size_t sketchIndex = (index + currentRead) * n;
        for (size_t j = 0; j < n; ++j) {
            size_t hashIndex = index * n * numKMers + j;
            kMer_t currentMin = ~(kMer_t) 0;
            for (size_t l = 0; l < numKMers; ++l) {
                kMer_t temp = hashes[hashIndex];
                hashIndex += n;
                currentMin = currentMin < temp ? currentMin : temp;
            }
            sketches[sketchIndex++] = currentMin;
        }
    }
}

NanoporeReads::~NanoporeReads() {
    hipFree(this->sketches);
}

void NanoporeReads::printHashes() {
    for (size_t i = 0; i < this->numReads; ++i) {
        std::cout << this->readPos[i];
        for (size_t j = 0; j < this->n; ++j) {
            std::cout << ", \"" << this->sketches[i * this->n + j] << "\"";
        }
        std::cout << std::endl;
    }
}