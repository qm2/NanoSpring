#include "hip/hip_runtime.h"
#include "../include/NanoporeReads.cuh"

NanoporeReads::NanoporeReads(char *fileName, int k, int n) : k(k), n(n), sketches(NULL) {
    std::ifstream infile(fileName);
    std::string line;
    numReads = 0;
    while (std::getline(infile, line)) {
//        std::cout << line << std::endl;
        size_t index = line.find(':');
        this->readPos.push_back(std::stol(line.substr(0, index)));
        {
            std::unique_ptr<std::string> ptr(new std::string(line.substr(index + 1)));
            this->editStrings.push_back(std::move(ptr));
        }
//        std::cout << this->readPos.back() << std::endl;
//        std::cout << this->editStrings.back() << std::endl;
        std::getline(infile, line);
        {
            std::unique_ptr<std::string> ptr(new std::string(line));
            this->readData.push_back(std::move(ptr));
        }
        numReads++;
    }
    this->readLen = this->readData[0]->length();
    std::cout << "numReads " << numReads << std::endl;
    std::cout << "readLen " << readLen << std::endl;
}

void NanoporeReads::calculateMinHashSketches() {
    // We store all the k-mers as uint64s. This would work for all k<=32,
    // which is definitely sufficient
    const size_t numKMers = this->readLen - this->k + 1;
    kMer_t *kMers;
    // Because of memory constraints on the GPUs we cannot deal with all the reads at once.
    // So we arrange the reads into blocks of blockSize reads and only work on a single block
    // at the same time.
    const size_t blockSize = 2048;
    std::cout << "numKMers " << numKMers << std::endl;

    hipMallocManaged(&(this->sketches), this->n * this->numReads * sizeof(kMer_t));

    std::random_device rd;
    std::mt19937_64 gen(rd());

    /* This is where you define the number generator for unsigned long long: */
    std::uniform_int_distribution<unsigned long long> dis;

    kMer_t *randNumbers;
    hipMallocManaged(&randNumbers, this->n * sizeof(kMer_t));
    for (size_t i = 0; i < this->n; ++i) {
        randNumbers[i] = dis(gen);
    }

    for (size_t currentRead = 0; currentRead < this->numReads; currentRead += blockSize) {
        std::cout << "CurrentRead " << currentRead << std::endl;
        const long readsLeft = numReads - (long) currentRead;
        const size_t currentBlockSize = readsLeft > blockSize ? blockSize : readsLeft;

        auto generateKMers = [&]() {
            hipMallocManaged(&kMers, currentBlockSize * numKMers * sizeof(kMer_t));

            for (size_t i = 0; i < currentBlockSize; i++) {
                size_t baseIndex = i * numKMers;
#pragma omp parallel for
                for (size_t index = baseIndex; index < numKMers + baseIndex; index++) {
                    kMers[index] =
                            kMerToInt(readData[i + currentRead]->substr(
                                    index - baseIndex, this->k));
                }
            }
        };

        generateKMers();

//        for (size_t i = 0; i < currentBlockSize * numKMers; ++i) {
//            std::cout << i << " " << kMers[i] << std::endl;
//        }

        // Now we generate all hashes
        // hashes is indexed by (read number, k-mer number, hash number)
        kMer_t *hashes;
        hipMallocManaged(&hashes, this->n * currentBlockSize * numKMers * sizeof(kMer_t));

        const size_t blockSize = 512;
        const size_t numBlocks = 512;
        hashKMer <<< numBlocks, blockSize >>>(currentBlockSize * numKMers,
                                              this->n, kMers, hashes, randNumbers);
        // Finish calculating the hashes and frees unneeded memory
        hipDeviceSynchronize();

//        for (size_t i = 0; i < currentBlockSize * numKMers * n; ++i) {
//            std::cout << i << " " << hashes[i] << std::endl;
//        }

        // Now we are going to compute the sketches which are the minimums of the hashes
        calcSketch
        <<< (currentBlockSize + blockSize - 1)
        / blockSize, blockSize >>>(currentBlockSize,
                                   currentRead, numKMers,
                                   this->n, hashes,
                                   this->sketches, kMers);
        hipDeviceSynchronize();
        hipFree(kMers);
        hipFree(hashes);
    }
    hipFree(randNumbers);
    populateHashTables();
}

kMer_t NanoporeReads::kMerToInt(const std::string &s) {
    size_t l = s.length();
    kMer_t result = 0;
    for (size_t i = 0; i < l; ++i) {
        result <<= 2;
        result |= baseToInt(s[i]);
    }
    return result;
}

char NanoporeReads::baseToInt(const char base) {
    switch (base) {
        case 'A':
            return 0;
        case 'T':
            return 1;
        case 'C':
            return 2;
        case 'G':
            return 3;
        default:
            std::cout << "Oh No!" << std::endl;
            return 0;
    }
}

__global__ void hashKMer(const size_t totalKMers, const size_t n,
                         kMer_t *kMers, kMer_t *hashes, kMer_t *randNumbers) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    for (size_t i = index; i < totalKMers; i += stride) {
        size_t hashIndex = i * n;
        kMer_t currentHash = kMers[i];
        currentHash = (currentHash * (uint64_t) HASH_C64);
        currentHash ^= randNumbers[0];
        hashes[hashIndex++] = currentHash;
        for (size_t j = 1; j < n; j++) {
            kMer_t newHash = ((currentHash >> ROTATE_BITS)
                              | (currentHash << (KMER_BITS - ROTATE_BITS)))
                             ^0xABCD32108475AC38;
            newHash = (newHash * (uint64_t) HASH_C64);
            newHash ^= randNumbers[j];
            newHash += currentHash;
            currentHash = newHash;
            hashes[hashIndex++] = currentHash;
        }
    }
}

__global__ void calcSketch(const size_t numReads, const size_t currentRead,
                           const size_t numKMers, const size_t n,
                           kMer_t *hashes, kMer_t *sketches, kMer_t *kMers) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    for (size_t i = index; i < numReads; i += stride) {
        size_t sketchIndex = (i + currentRead) * n;
        for (size_t j = 0; j < n; ++j) {
            size_t hashIndex = i * n * numKMers + j;
            kMer_t currentMin = ~(kMer_t) 0;
            size_t minIndex = 0;
            for (size_t l = 0; l < numKMers; ++l) {
                kMer_t temp = hashes[hashIndex];
                hashIndex += n;
                minIndex = currentMin < temp ? minIndex : l;
                currentMin = currentMin < temp ? currentMin : temp;
            }
            //std::cout << "thread: " << i << " hash id: " << j << std::endl;
            //std::cout << "minIndex " << minIndex << ":" << kMers[i * numKMers + minIndex];

            if (kMers)
                sketches[sketchIndex++] = kMers[i * numKMers + minIndex];
            else
                sketches[sketchIndex++] = currentMin;
        }
    }
}

NanoporeReads::~NanoporeReads() {
    hipFree(this->sketches);
}

void NanoporeReads::printHashes() {
    for (size_t i = 0; i < this->numReads; ++i) {
        std::cout << this->readPos[i];
        for (size_t j = 0; j < this->n; ++j) {
            std::cout << ", \"" << this->sketches[i * this->n + j] << "\"";
        }
        std::cout << std::endl;
    }
}

void NanoporeReads::populateHashTables() {
    std::cout << "Starting to populate hash tables" << std::endl;
    auto start = std::chrono::high_resolution_clock::now();
    for (size_t i = 0; i < this->n; ++i) {
        this->hashTables.push_back(std::map<kMer_t, std::vector<size_t>>());
    }
#pragma omp parallel for
    for (size_t i = 0; i < this->n; ++i) {
        std::map<kMer_t, std::vector<size_t>> &hT = this->hashTables[i];
        size_t currentIndex = i;
        for (size_t j = 0; j < this->numReads; ++j) {
            currentIndex += this->n;
            try {
                hT[this->sketches[currentIndex]].push_back(j);
            } catch (std::out_of_range) {
                std::vector<size_t> v;
                hT[this->sketches[currentIndex]] = v;
                v.push_back(j);
            }
        }
    }
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
    std::cout << "finished populating hash tables" << std::endl;
    std::cout << duration.count() << " milliseconds passed" << std::endl;
}