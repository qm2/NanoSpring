#include "hip/hip_runtime.h"
#include "../include/NanoporeReads.cuh"

NanoporeReads::NanoporeReads(const char *fileName, int k, int n) : k(k), n(n), sketches(NULL) {
    std::ifstream infile(fileName);
    std::string line;
    numReads = 0;
    while (std::getline(infile, line)) {
//        std::cout << line << std::endl;
        size_t index = line.find(':');
        readPos.push_back(std::stol(line.substr(0, index)));
        {
            std::unique_ptr<std::string> ptr(new std::string(line.substr(index + 1)));
            editStrings.push_back(std::move(ptr));
        }
//        std::cout << readPos.back() << std::endl;
//        std::cout << editStrings.back() << std::endl;
        std::getline(infile, line);
        {
            std::unique_ptr<std::string> ptr(new std::string(line));
            readData.push_back(std::move(ptr));
        }
        numReads++;
    }
    readLen = readData[0]->length();
    readPosSorted = readPos;
    std::sort(readPosSorted.begin(), readPosSorted.end());
    std::cout << "numReads " << numReads << std::endl;
    std::cout << "readLen " << readLen << std::endl;
}

void NanoporeReads::calculateMinHashSketches() {
    // We store all the k-mers as uint64s. This would work for all k<=32,
    // which is definitely sufficient
    const size_t numKMers = readLen - k + 1;
    kMer_t *kMers;
    // Because of memory constraints on the GPUs we cannot deal with all the reads at once.
    // So we arrange the reads into blocks of blockSize reads and only work on a single block
    // at the same time.
    const size_t blockSize = 1024;
    std::cout << "numKMers " << numKMers << std::endl;

    hipMallocManaged(&(sketches), n * numReads * sizeof(kMer_t));

    std::random_device rd;
    std::mt19937_64 gen(rd());

    /* This is where you define the number generator for unsigned long long: */
    std::uniform_int_distribution<unsigned long long> dis;

    kMer_t *randNumbers;
    hipMallocManaged(&randNumbers, n * sizeof(kMer_t));
    for (size_t i = 0; i < n; ++i) {
        randNumbers[i] = dis(gen);
    }

    for (size_t currentRead = 0; currentRead < numReads; currentRead += blockSize) {
        std::cout << "CurrentRead " << currentRead << std::endl;
        const long readsLeft = numReads - (long) currentRead;
        const size_t currentBlockSize = readsLeft > blockSize ? blockSize : readsLeft;

        auto generateKMers = [&]() {
            hipMallocManaged(&kMers, currentBlockSize * numKMers * sizeof(kMer_t));

            for (size_t i = 0; i < currentBlockSize; i++) {
                size_t baseIndex = i * numKMers;
#pragma omp parallel for
                for (size_t index = baseIndex; index < numKMers + baseIndex; index++) {
                    kMers[index] =
                            kMerToInt(readData[i + currentRead]->substr(
                                    index - baseIndex, k));
                }
            }
        };

        generateKMers();

//        for (size_t i = 0; i < currentBlockSize * numKMers; ++i) {
//            std::cout << i << " " << kMers[i] << std::endl;
//        }

        // Now we generate all hashes
        // hashes is indexed by (read number, k-mer number, hash number)
        kMer_t *hashes;
        hipMallocManaged(&hashes, n * currentBlockSize * numKMers * sizeof(kMer_t));
#ifdef _GPU
        const size_t blockSize = 512;
        const size_t numBlocks = 512;
        hashKMer_GPU <<< numBlocks, blockSize >>>(currentBlockSize * numKMers,
                                              n, kMers, hashes, randNumbers);
        // Finish calculating the hashes and frees unneeded memory
        hipDeviceSynchronize();
#else
        {
            auto start = std::chrono::high_resolution_clock::now();
            hashKMer(currentBlockSize, numKMers, n, kMers, hashes, randNumbers);
            auto end = std::chrono::high_resolution_clock::now();
            auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
            std::cout << "finished hashKMer" << std::endl;
            std::cout << duration.count() << " milliseconds passed" << std::endl;
        }
#endif

//        for (size_t i = 0; i < currentBlockSize * numKMers * n; ++i) {
//            std::cout << i << " " << hashes[i] << std::endl;
//        }

        // Now we are going to compute the sketches which are the minimums of the hashes
#ifdef _GPU
        calcSketch_GPU
        <<< (currentBlockSize + blockSize - 1)
        / blockSize, blockSize >>>(currentBlockSize,
                                   currentRead, numKMers,
                                   n, hashes,
                                   sketches, kMers);
        hipDeviceSynchronize();
#else
        {
            auto start = std::chrono::high_resolution_clock::now();
            calcSketch(currentBlockSize, currentRead, numKMers, n, hashes,
                       sketches, kMers);
            auto end = std::chrono::high_resolution_clock::now();
            auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
            std::cout << "finished calcSketch" << std::endl;
            std::cout << duration.count() << " milliseconds passed" << std::endl;
        }
#endif
        hipFree(kMers);
        hipFree(hashes);
    }
    hipFree(randNumbers);
    populateHashTables();
}

kMer_t NanoporeReads::kMerToInt(const std::string &s) {
    size_t l = s.length();
    kMer_t result = 0;
    for (size_t i = 0; i < l; ++i) {
        result <<= 2;
        result |= baseToInt(s[i]);
    }
    return result;
}

// Using the bit operations version of this function provides a 13X improvement in speed
char NanoporeReads::baseToInt(const char base) {
    return (base & 0b10) | ((base & 0b100) >> 2);
//    switch (base) {
//        case 'A':
//            return 0;
//        case 'T':
//            return 1;
//        case 'C':
//            return 2;
//        case 'G':
//            return 3;
//        default:
//            std::cout << "Oh No!" << std::endl;
//            return 0;
//    }
}

void
NanoporeReads::hashKMer(const size_t numReads, const size_t numKMers, const size_t n, kMer_t *kMers, kMer_t *hashes,
                        kMer_t *randNumbers) {
#pragma omp parallel for
    for (size_t i = 0; i < numReads; i += 1) {
//#pragma omp parallel for
        for (size_t j = 0; j < numKMers; j += 1) {
            size_t hashIndex = i * n * numKMers + j * n;
            kMer_t currentHash = kMers[i * numKMers + j];
            currentHash = (currentHash * (uint64_t) HASH_C64);
            currentHash ^= randNumbers[0];
            hashes[hashIndex] = currentHash;
            for (size_t l = 1; l < n; l++) {
                kMer_t newHash = ((currentHash >> ROTATE_BITS)
                                  | (currentHash << (KMER_BITS - ROTATE_BITS)))
                                 ^0xABCD32108475AC38;
                newHash = (newHash * (uint64_t) HASH_C64);
                newHash ^= randNumbers[l];
                newHash += currentHash;
                currentHash = newHash;
                hashes[hashIndex + l] = currentHash;
            }
        }
    }
}

__global__ void hashKMer_GPU(const size_t totalKMers, const size_t n,
                             kMer_t *kMers, kMer_t
                             *hashes,
                             kMer_t *randNumbers) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    for (
            size_t i = index;
            i < totalKMers;
            i += stride) {
        size_t hashIndex = i * n;
        kMer_t currentHash = kMers[i];
        currentHash = (currentHash * (uint64_t) HASH_C64);
        currentHash ^= randNumbers[0];
        hashes[hashIndex++] =
                currentHash;
        for (
                size_t j = 1;
                j < n;
                j++) {
            kMer_t newHash = ((currentHash >> ROTATE_BITS)
                              | (currentHash << (KMER_BITS - ROTATE_BITS)))
                             ^0xABCD32108475AC38;
            newHash = (newHash * (uint64_t) HASH_C64);
            newHash ^= randNumbers[j];
            newHash += currentHash;
            currentHash = newHash;
            hashes[hashIndex++] =
                    currentHash;
        }
    }
}

void NanoporeReads::calcSketch(const size_t numReads, const size_t currentRead,
                               const size_t numKMers, const size_t n,
                               kMer_t *hashes, kMer_t *sketches, kMer_t *kMers) {
//#pragma omp parallel for
    for (size_t i = 0; i < numReads; i++) {
        size_t sketchIndex = (i + currentRead) * n;
#pragma omp parallel for
        for (size_t j = 0; j < n; ++j) {
            size_t hashIndex = i * n * numKMers + j;
            kMer_t currentMin = ~(kMer_t) 0;
//            size_t minIndex = 0;
//#pragma omp parallel for reduction(min:currentMin)
            for (size_t l = 0; l < numKMers * n; l += n) {
//                kMer_t temp = hashes[hashIndex];
//                hashIndex += n;
                kMer_t temp = hashes[hashIndex + l];
//                minIndex = currentMin < temp ? minIndex : l;
                currentMin = currentMin < temp ? currentMin : temp;
            }
//            if (kMers) {
//                sketches[sketchIndex++] = kMers[i * numKMers + minIndex];
//            } else
            sketches[sketchIndex + j] = currentMin;
        }
    }
}

__global__ void calcSketch_GPU(const size_t numReads, const size_t currentRead,
                               const size_t numKMers, const size_t n,
                               kMer_t *hashes, kMer_t *sketches, kMer_t *kMers) {

    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    for (size_t i = index; i < numReads; i += stride) {
        size_t sketchIndex = (i + currentRead) * n;
        for (size_t j = 0; j < n; ++j) {
            size_t hashIndex = i * n * numKMers + j;
            kMer_t currentMin = ~(kMer_t) 0;
            size_t minIndex = 0;
            for (size_t l = 0; l < numKMers; ++l) {
                kMer_t temp = hashes[hashIndex];
                hashIndex += n;
                minIndex = currentMin < temp ? minIndex : l;
                currentMin = currentMin < temp ? currentMin : temp;
//                currentMin = temp ^ ((currentMin ^ temp) & -(currentMin < temp));
            }
            //std::cout << "thread: " << i << " hash id: " << j << std::endl;
            //std::cout << "minIndex " << minIndex << ":" << kMers[i * numKMers + minIndex];

            if (kMers) {
                sketches[sketchIndex++] = kMers[i * numKMers + minIndex];
            } else
                sketches[sketchIndex++] = currentMin;
        }
    }
}

NanoporeReads::~NanoporeReads() {
    hipFree(sketches);
}

void NanoporeReads::printHashes() {
    for (size_t i = 0; i < numReads; ++i) {
        std::cout << readPos[i];
        for (size_t j = 0; j < n; ++j) {
            std::cout << ", \"" << sketches[i * n + j] << "\"";
        }
        std::cout << std::endl;
    }
}

void NanoporeReads::populateHashTables() {
    std::cout << "Starting to populate hash tables" << std::endl;
    auto start = std::chrono::high_resolution_clock::now();
    for (size_t i = 0; i < n; ++i) {
        hashTables.push_back(std::map<kMer_t, std::vector<size_t >>());
    }
//#pragma omp parallel for
    for (size_t i = 0; i < n; ++i) {
        std::map<kMer_t, std::vector<size_t>> &hT = hashTables[i];
        size_t currentIndex = i;
        for (size_t j = 0; j < numReads; ++j) {
            hT[sketches[currentIndex]].push_back(j);
//            std::cout << "Inserting " << j << " to " << sketches[currentIndex] << std::endl;
            currentIndex += n;
        }
    }

    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
    std::cout << "finished populating hash tables" << std::endl;
    std::cout << duration.count() << " milliseconds passed" << std::endl;
}

filterStats NanoporeReads::getFilterStats(unsigned int overlapBaseThreshold, unsigned int overlapSketchThreshold) {
    filterStats result(overlapBaseThreshold, overlapSketchThreshold);
    // First we calculate the number of overlaps and disjoints
    for (size_t i = 0; i < numReads; ++i) {
        long curTh = readPosSorted[i] + readLen - overlapBaseThreshold;
        for (size_t j = i + 1; j < numReads; ++j) {
            if (((long) readPosSorted[j]) <= curTh)
                result.numOverlaps++;
            else
                break;
        }
    }
    result.numDisjoint = (numReads * (unsigned long long) (numReads - 1)) / 2 - result.numOverlaps;

    // Now we calculate falsePositives, falseNegatives, etc
    for (size_t i = 0; i < numReads; ++i) {
        std::multiset<size_t> matches;
        unsigned long curPos = readPos[i];
        long th = readLen - overlapBaseThreshold;
        for (size_t sketchIndex = 0; sketchIndex < n; ++sketchIndex) {
            kMer_t curHash = sketches[i * n + sketchIndex];
//            std::cout << i << " " << sketchIndex << " " << curHash << std::endl;
//            auto currentMap = hashTables[sketchIndex];
//            for (auto p : currentMap) {
//                std::cout << p.first << " ";
//            }
//            std::cout << std::endl;
            std::vector<size_t> &m = hashTables[sketchIndex].at(curHash);
            matches.insert(m.begin(), m.end());
        }
        auto end = matches.end();
        for (auto it = matches.begin(); it != end; it = matches.upper_bound(*it)) {
            if (*it <= i)
                continue;
            unsigned long pos = readPos[*it];
            if (matches.count(*it) >= overlapSketchThreshold) {
                if (abs((long) pos - (long) curPos) > th)
                    result.falsePositives++;
                result.totalPositive++;
            }
        }
//        std::cout << std::endl;
    }

    result.totalNegative = result.numOverlaps + result.numDisjoint - result.totalPositive;
    result.falseNegatives = result.numOverlaps - result.totalPositive + result.falsePositives;
    return result;
}

filterStats::filterStats(unsigned int overlapBaseThreshold, unsigned int overlapSketchThreshold) :
        overlapBaseThreshold(overlapBaseThreshold), overlapSketchThreshold(overlapSketchThreshold),
        totalPositive(0), totalNegative(0), numOverlaps(0), numDisjoint(0),
        falsePositives(0), falseNegatives(0) {
}

std::ostream &operator<<(std::ostream &out, const filterStats &o) {
    const int w = 13;
    out << std::setw(w) << "overlapBaseTh" << ","
        << std::setw(w) << "numKMerTh" << ","
        << std::setw(w) << "totalPos" << ","
        << std::setw(w) << "totalNeg" << ","
        << std::setw(w) << "numOverlaps" << ","
        << std::setw(w) << "numDisjoint" << ","
        << std::setw(w) << "falsePos" << ","
        << std::setw(w) << "falseNeg" << std::endl;
    out << std::setw(w) << o.overlapBaseThreshold << ","
        << std::setw(w) << o.overlapSketchThreshold << ","
        << std::setw(w) << o.totalPositive << ","
        << std::setw(w) << o.totalNegative << ","
        << std::setw(w) << o.numOverlaps << ","
        << std::setw(w) << o.numDisjoint << ","
        << std::setw(w) << o.falsePositives << ","
        << std::setw(w) << o.falseNegatives << std::endl;
    return out;
}

MinHashReadFilter::MinHashReadFilter(size_t overlapSketchThreshold, NanoporeReads &nR) :
        overlapSketchThreshold(overlapSketchThreshold), nR(nR) {
}

void MinHashReadFilter::getFilteredReads(size_t readToFind, std::vector<size_t> &results) {
    size_t n = nR.n;
    auto &sketches = nR.sketches;
    auto &hashTables = nR.hashTables;
    std::vector<size_t> matches;
    results.clear();
    for (size_t sketchIndex = 0; sketchIndex < n; ++sketchIndex) {
        kMer_t curHash = sketches[readToFind * n + sketchIndex];
        std::vector<size_t> &m = hashTables[sketchIndex].at(curHash);
        matches.insert(matches.end(), m.begin(), m.end());
    }
    std::sort(matches.begin(), matches.end());
    auto end = matches.end();
    auto next = matches.begin();
    for (auto it = matches.begin(); it != end; it = next) {
        next = std::upper_bound(it, end, *it);
        if (*it == readToFind)
            continue;

        if (next - it >= overlapSketchThreshold) {
            results.push_back(*it);
        }
    }
}